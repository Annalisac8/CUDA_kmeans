﻿#include "hip/hip_runtime.h"
#include ""
#include "kernel.cuh"

#include <stdio.h>

#include <cmath>
#include <iostream>

// variabili globali utilizzate nei kernel CUDA
// GLOBALI = risiedono nella memoria costante della GPU
// accessibili a tutti i thread efficentemente 
__constant__ short constNumCluster;
__constant__ int constNumPunti;
__constant__ short constDimPunti;

//Funzione di controllo degli errori CUDA
void ControllaErroreCuda(const char* file, unsigned linea, const char* istruzione, hipError_t errore) {
    if (errore == hipSuccess) {
        return;
    }
    std::cerr << istruzione << " ha restituito " << hipGetErrorString(errore) << "(" << errore << ") in " << file << ":" << linea << std::endl;
    exit(1);
}
