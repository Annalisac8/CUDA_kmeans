#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>

/*
__global__ void assign_clusters(double* points, double* centroids, int* assignments, int numPoints, int numCentroids, int dimensions) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPoints) return;

    //printf("Thread %d (idx %d): Inizio assegnazione cluster.\n", threadIdx.x, idx);

    double minDist = 1e20;
    int bestCluster = 0;

    for (int c = 0; c < numCentroids; c++) {
        double dist = 0.0;
        for (int d = 0; d < dimensions; d++) {
            double diff = points[idx * dimensions + d] - centroids[c * dimensions + d];
            dist += diff * diff;
        }
        
        if (dist < minDist) {
            minDist = dist;
            bestCluster = c;
        }
        
    }
    if (idx >= numPoints) {
        //printf("Errore: Thread %d, idx %d fuori dai limiti (numPoints: %d)\n", threadIdx.x, idx, numPoints);
        return;
    }
    assignments[idx] = bestCluster;

    //printf("Thread %d (idx %d): Assegnato al cluster %d.\n", threadIdx.x, idx, bestCluster);
}
*/
/*
__global__ void update_centroids(double* points, double* centroids, int* assignments, int numPoints, int numCentroids, int dimensions) {
    extern __shared__ double sharedMem[];
    double* sums = sharedMem;
    int* clusterCounts = (int*)&sums[numCentroids * dimensions];

    int centroidIdx = blockIdx.x;
    if (centroidIdx >= numCentroids) return;

    for (int d = threadIdx.x; d < dimensions; d += blockDim.x) {
        sums[centroidIdx * dimensions + d] = 0.0f;
    }

    if (threadIdx.x == 0) clusterCounts[centroidIdx] = 0;

    __syncthreads();

    for (int i = threadIdx.x; i < numPoints; i += blockDim.x) {
        if (assignments[i] == centroidIdx) {
            for (int d = 0; d < dimensions; d++) {
                atomicAdd(&sums[centroidIdx * dimensions + d], points[i * dimensions + d]);
            }
            atomicAdd(&clusterCounts[centroidIdx], 1);
        }
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        for (int d = 0; d < dimensions; d++) {
            if (clusterCounts[centroidIdx] > 0)
                centroids[centroidIdx * dimensions + d] = sums[centroidIdx * dimensions + d] / clusterCounts[centroidIdx];
        }

        //printf("Centroide %d aggiornato: ", centroidIdx);
        for (int d = 0; d < dimensions; d++) {
            //printf("%f ", centroids[centroidIdx * dimensions + d]);
        }
        //printf("\n");
    }
}
*/
#include <hiprand/hiprand_kernel.h> // Inclusione di cuRAND per generazione numeri casuali


/*
__global__ void update_centroids(double* points, double* centroids, int* assignments, int numPoints, int numCentroids, int dimensions, unsigned long seed) {
    extern __shared__ double sharedMem[];
    double* sums = sharedMem;
    int* clusterCounts = (int*)&sums[numCentroids * dimensions];
    


    int centroidIdx = blockIdx.x;
    if (centroidIdx >= numCentroids) return;

    // Inizializzazione memoria condivisa
    for (int d = threadIdx.x; d < dimensions; d += blockDim.x) {
        sums[centroidIdx * dimensions + d] = 0.0f;
    }
    if (threadIdx.x == 0) {
        clusterCounts[centroidIdx] = 0;
    }

    __syncthreads();

    // Sommare le coordinate dei punti assegnati a questo centroide
    for (int i = threadIdx.x; i < numPoints; i += blockDim.x) {
        if (assignments[i] == centroidIdx) {
            for (int d = 0; d < dimensions; d++) {
                atomicAdd(&sums[centroidIdx * dimensions + d], points[i * dimensions + d]);
            }
            atomicAdd(&clusterCounts[centroidIdx], 1);
        }
    }

    __syncthreads();


    // Un thread per blocco aggiorna il centroide
    if (threadIdx.x == 0) {
        if (clusterCounts[centroidIdx] > 0) {
            for (int d = 0; d < dimensions; d++) {
                centroids[centroidIdx * dimensions + d] = sums[centroidIdx * dimensions + d] / clusterCounts[centroidIdx];
            }
        }
        else {
            // Se un cluster è vuoto, assegniamo un punto casuale usando cuRAND
            hiprandState state;
            hiprand_init(seed + centroidIdx, 0, 0, &state);
            int randomIdx = hiprand(&state) % numPoints;

            for (int d = 0; d < dimensions; d++) {
                centroids[centroidIdx * dimensions + d] = points[randomIdx * dimensions + d];
            }
            //printf("Centroide %d era vuoto e riassegnato al punto %d\n", centroidIdx, randomIdx);
        }
        //printf("Centroide %d aggiornato con %d punti\n", centroidIdx, clusterCounts[centroidIdx]);
    }
}


*/

/*

 
void kmeans_cuda(double* d_points, double* d_centroids, int* d_assignments, int numPoints, int numCentroids, int dimensions, int maxIterations, double tolerance, std::vector<double>& h_oldCentroids, std::vector<double>& h_currentCentroids) {

    if (d_points == nullptr || d_centroids == nullptr || d_assignments == nullptr) {
        //printf("Errore: Puntatori GPU non validi.\n");
        return;
    }

    //printf("Inizio kmeans_cuda (numPoints: %d, numCentroids: %d, dimensions: %d)\n", numPoints, numCentroids, dimensions);

    // Allocazione memoria per i centroidi precedenti
    double* d_oldCentroids;
    //printf("Allocazione d_oldCentroids (numCentroids: %d, dimensions: %d)...\n", numCentroids, dimensions);
    CUDA_CHECK(hipMalloc(&d_oldCentroids, numCentroids * dimensions * sizeof(double)));
    //printf("Allocazione d_oldCentroids completata con successo.\n");

    // Configurazione per i kernel
    dim3 threadsPerBlock(128); //128
    dim3 blocksPerGrid((numPoints + threadsPerBlock.x - 1) / threadsPerBlock.x);
    size_t sharedMemSize = sizeof(double) * dimensions * threadsPerBlock.x + sizeof(int) * threadsPerBlock.x;

    //printf("Memoria condivisa allocata per blocco: %lu bytes\n", sharedMemSize);

    for (int iter = 0; iter < maxIterations; iter++) {


        //printf("Inizio iterazione %d\n", iter);

        // Copia d_centroids in d_oldCentroids
        CUDA_CHECK(hipMemcpy(d_oldCentroids, d_centroids, numCentroids * dimensions * sizeof(double), hipMemcpyDeviceToDevice));
        //printf("Copia di d_centroids in d_oldCentroids completata con successo.\n");

        // Kernel per assegnare i punti ai cluster
        assign_clusters << <blocksPerGrid, threadsPerBlock >> > (d_points, d_centroids, d_assignments, numPoints, numCentroids, dimensions);
        CUDA_CHECK(hipDeviceSynchronize());
        //printf("Kernel assign_clusters completato senza errori.\n");

        // Kernel per aggiornare i centroidi
        //update_centroids << <numCentroids, threadsPerBlock, sharedMemSize >> > (d_points, d_centroids, d_assignments, numPoints, numCentroids, dimensions);
        //CUDA_CHECK(hipDeviceSynchronize());
        //printf("Kernel update_centroids completato senza errori.\n");
        
        unsigned long seed = time(NULL); // Definiamo un seed casuale per cuRAND
        int sharedMemorySize = numCentroids * dimensions * sizeof(double) + numCentroids * sizeof(int);

        //printf("Memoria condivisa allocata: %zu bytes\n", sharedMemSize);


        update_centroids << <numCentroids, 256, sharedMemorySize >> > (d_points, d_centroids,d_assignments, numPoints, numCentroids, dimensions, seed);
        

        CUDA_CHECK(hipDeviceSynchronize());
        


        // Copia dei dati dalla GPU per la verifica della convergenza
        std::vector<double> h_oldCentroids(numCentroids * dimensions);
        CUDA_CHECK(hipMemcpy(h_oldCentroids.data(), d_oldCentroids, numCentroids * dimensions * sizeof(double), hipMemcpyDeviceToHost));
        //printf("Copia di d_oldCentroids nella memoria host completata con successo.\n");

        std::vector<double> h_currentCentroids(numCentroids * dimensions);
        CUDA_CHECK(hipMemcpy(h_currentCentroids.data(), d_centroids, numCentroids * dimensions * sizeof(double), hipMemcpyDeviceToHost));
        // printf("Copia di d_centroids nella memoria host completata con successo.\n");

        
        double maxChange = 0.0;
        for (int i = 0; i < numCentroids * dimensions; i++) {
            double change = fabs(h_currentCentroids[i] - h_oldCentroids[i]);
            if (change > maxChange) maxChange = change;
        }
        if (maxChange < tolerance) {
            printf("Convergenza Cuda raggiunta dopo %d iterazioni.\n", iter + 1);
            break;
        }
        

         //printf("Iterazione %d completata senza errori.\n", iter);
    }

    // Rilascio della memoria GPU
    //printf("Rilascio della memoria GPU in kmeans_cuda...\n");
    if (d_oldCentroids) {
        CUDA_CHECK(hipFree(d_oldCentroids));
        //printf("Memoria d_oldCentroids rilasciata con successo.\n");
    }

    //printf("kmeans_cuda completato senza errori.\n");
}
*/#include "kernel.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <device_atomic_functions.h>
#include <cmath>
#include <iostream>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__global__ void assignClusters(const double* points, const double* centroids, int* assignments,
    int numPoints, int numCentroids, int dimensions) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPoints) return;

    double minDist = INFINITY;
    int bestCluster = -1;

    for (int c = 0; c < numCentroids; ++c) {
        double dist = 0.0;
        for (int d = 0; d < dimensions; ++d) {
            double diff = points[idx * dimensions + d] - centroids[c * dimensions + d];
            dist += diff * diff;
        }
        dist = sqrt(dist);
        if (dist < minDist) {
            minDist = dist;
            bestCluster = c;
        }
    }
    assignments[idx] = bestCluster;
   // printf("Punto %d assegnato al cluster %d con distanza %.4f", idx, bestCluster, minDist);
}

__global__ void updateCentroids(const double* points, double* newCentroids, int* assignments,
    int numPoints, int numCentroids, int dimensions, int* clusterSizes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPoints) return;

    int cluster = assignments[idx];
    if (cluster == -1) return;

    for (int d = 0; d < dimensions; ++d) {
        atomicAdd(&newCentroids[cluster * dimensions + d], points[idx * dimensions + d]);
    }
    atomicAdd(&clusterSizes[cluster], 1);
}

__global__ void normalizeCentroids(double* centroids, double* newCentroids, int* clusterSizes, int numCentroids, int dimensions) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numCentroids) return;

    int size = clusterSizes[idx];
    if (size > 0) {
        for (int d = 0; d < dimensions; ++d) {
            centroids[idx * dimensions + d] = newCentroids[idx * dimensions + d] / size;
        }
    }
}

void kmeans_cuda(double* d_points, double* d_centroids, int* d_assignments,
    int numPoints, int numCentroids, int dimensions, int maxIters, double tol,
    std::vector<double>& h_oldCentroids, std::vector<double>& h_currentCentroids) {
    int* d_clusterSizes;
    double* d_newCentroids;
    CUDA_CHECK(hipMalloc(&d_clusterSizes, numCentroids * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_newCentroids, numCentroids * dimensions * sizeof(double)));

    int blockSize = 256;
    int gridSizePoints = (numPoints + blockSize - 1) / blockSize;
    int gridSizeCentroids = (numCentroids + blockSize - 1) / blockSize;


    bool convergenza=false;

    //for (int iter = 0; iter < maxIters; ++iter) {
    int iter = 0;
    while(!convergenza){
        CUDA_CHECK(hipMemset(d_clusterSizes, 0, numCentroids * sizeof(int)));
        CUDA_CHECK(hipMemset(d_newCentroids, 0, numCentroids * dimensions * sizeof(double)));

        assignClusters << <gridSizePoints, blockSize >> > (d_points, d_centroids, d_assignments, numPoints, numCentroids, dimensions);
        CUDA_CHECK(hipDeviceSynchronize());

        updateCentroids << <gridSizePoints, blockSize >> > (d_points, d_newCentroids, d_assignments, numPoints, numCentroids, dimensions, d_clusterSizes);
        CUDA_CHECK(hipDeviceSynchronize());

        normalizeCentroids << <gridSizeCentroids, blockSize >> > (d_centroids, d_newCentroids, d_clusterSizes, numCentroids, dimensions);
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(hipMemcpy(h_currentCentroids.data(), d_centroids, numCentroids * dimensions * sizeof(double), hipMemcpyDeviceToHost));

        /*
        std::cout << "\n--- [DEBUG] Centroidi aggiornati cuda ***\n";
        for (int c = 0; c < numCentroids; c++) {
            std::cout << "Centroide " << c << ": (";
            for (int d = 0; d < dimensions; d++) {
                std::cout << h_currentCentroids[c * dimensions + d];
                if (d < dimensions - 1) std::cout << ", ";
            }
            std::cout << ")\n";

        }
        */
/*
        bool converged = true;
        for (int c = 0; c < numCentroids; ++c) {
            for (int d = 0; d < dimensions; ++d) {
                if (h_currentCentroids[c * dimensions + d] != h_oldCentroids[c * dimensions + d]) {
                    converged = false;
                    break;
                }
            }
            if (!converged) break;
        }

        if (converged) {
            printf("Convergenza raggiunta dopo %d iterazioni\n", iter + 1);
            break;
        }
  */      

        for (int c = 0; c < numCentroids; ++c) {
            double shift = 0.0;
            convergenza = false;

            for (int d = 0; d < dimensions; ++d) {
                double diff = h_currentCentroids[c * dimensions + d] - h_oldCentroids[c * dimensions + d];
                shift += diff * diff;
            }
            //maxShift = fmax(maxShift, sqrt(shift));

            if ((std::sqrt(shift) / dimensions) > tol) {
                //printf("Non convergente\n");
                break;
            }
            else {
                convergenza = true;
            }
        }

        iter++;
        h_oldCentroids = h_currentCentroids;

    }

    // Stampa il numero totale di iterazioni eseguite
    std::cout << "Numero di iterazioni per convergenza: " << iter << " \n";

    CUDA_CHECK(hipFree(d_clusterSizes));
    CUDA_CHECK(hipFree(d_newCentroids));
}
